/*
Katelyn Jaing
Hector Medina
CPSC 479-01
Homework 6:
	Problem 1: compute square of N dimensional matrix (N=32)
				using a host function
		nvcc <filename>
		./a.out
	Problem 2: Fortran 90 program
		Declare 4x4 matrix A (all values set to 5)
		Declare 4x4 matrix B (B(i,j) = i+j)
		Compute C = A - B
		Replace in C elements with value 8 to value 18
	
		gfortran <filename>
		./a.out
*/
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 32

__global__ void square( unsigned *h_matrix ){
        unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
        h_matrix[id] = id * id;
}

int main() {

        dim3 block(N, N);
        //creating the matrix variables
        unsigned *matrix, *h_matrix;
        //initializing the variables
        matrix = (unsigned *)malloc(N * N * sizeof(unsigned));
        hipMalloc(&h_matrix, N * N * sizeof(unsigned));
        //squaring the matrix
        square<<<1, block>>>(h_matrix);
        //copying the memory over
        hipMemcpy( matrix, h_matrix, N * N * sizeof(unsigned), hipMemcpyDeviceToHost );
        //print out the matrix
        for( int i = 0; i < N; i++ ){
                for( int j = 0; j < N; j++ ){
                        printf( "%d\t", matrix[ i * N + j ] );
                }
                printf( "\n" );
        }

        return 0;
}


















