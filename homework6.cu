
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 32

__global__ void square( unsigned *h_matrix ){
        unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
        h_matrix[id] = id * id;
}

int main() {

        dim3 block(N, N);
        //creating the matrix variables
        unsigned *matrix, *h_matrix;
        //initializing the variables
        matrix = (unsigned *)malloc(N * N * sizeof(unsigned));
        hipMalloc(&h_matrix, N * N * sizeof(unsigned));
        //squaring the matrix
        square<<<1, block>>>(h_matrix);
        //copying the memory over
        hipMemcpy( matrix, h_matrix, N * N * sizeof(unsigned), hipMemcpyDeviceToHost );
        //print out the matrix
        for( int i = 0; i < N; i++ ){
                for( int j = 0; j < N; j++ ){
                        printf( "%d\t", matrix[ i * N + j ] );
                }
                printf( "\n" );
        }

        return 0;
}


















