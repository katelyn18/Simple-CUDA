/*
Problem 1: initialize array of size 32 to 0
Problem 2: change array size to 1024
Problem 3: create another kernel that adds i to array[ i ]
Problem 4: change array size 8000 (check answer to Problem 3 still works)
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

//initialize array to 0
__global__ void kernel1( int N, int *d_array  ){
	for( int i = 0; i < N; i++ ){
		d_array[ i ] = 0;
	}
}

//add i to array[ i ]
__global__ void kernel2( int N, int *d_array ){
	for( int i = 0; i < N; i++ ){
		d_array[ i ] = i;
	}
}

int main(){
	
	int *array, *d_array; 
//DO NOT COMMENT OUT ANYTHING ABOVE THIS LINE


/*
	printf( "**********PROBLEM 1**********\n" );
	int N = 32; 
	array = (int*)malloc( N*sizeof(int) );
	cudaMalloc( &d_array, N*sizeof( int ) );

	kernel1<<<8, 128>>>( N, d_array );
	cudaMemcpy( array, d_array, N*sizeof(int), cudaMemcpyDeviceToHost );

	for( int i = 0; i < N; i++ ){
		printf( "array[ %d ] = %d ", i, array[ i ] );
		if( (i+1) % 4 == 0 ){
			printf( "\n" );
		}
	}
	//**********end problem 1**********
*/

/*
	printf( "**********PROBLEM 2**********\n" );
	int N2 = 1024;
	array = (int*)malloc( N2*sizeof(int) );
	cudaMalloc( &d_array, N2*sizeof(int) );

	kernel1<<<8, 128>>>( N2, d_array );
	cudaMemcpy( array, d_array, N2*sizeof(int), cudaMemcpyDeviceToHost );

	for( int i = 0; i < N2; i++ ){
		printf( "array[ %d ] = %d ", i, array[ i ] );
		if( (i+1) % 8 == 0 ){
			printf( "\n" );
		}
	}
	//**********end problem 2**********
*/

/*
	printf( "**********PROBLEM 3**********\n" );
	int N2 = 1024;
	array = (int*)malloc( N2*sizeof(int) );
	cudaMalloc( &d_array, N2*sizeof(int) );

	kernel2<<<8, 128>>>( N2, d_array );
	cudaMemcpy( array, d_array, N2*sizeof(int), cudaMemcpyDeviceToHost );

	for( int i = 0; i < N2; i++ ){
		printf( "array[ %d ] = %d ", i, array[ i ] );
		if( (i+1) % 8 == 0 ){
			printf( "\n" );
		}
	}
	//**********end problem 3**********
*/

	printf( "**********PROBLEM 4**********\n" );
	int N3 = 8000; 
	array = (int*)malloc( N3*sizeof(int) );
	hipMalloc( &d_array, N3*sizeof(int) );

	kernel2<<<8, 128>>>( N3, d_array );
	hipMemcpy( array, d_array, N3*sizeof(int), hipMemcpyDeviceToHost );

	for( int i = 0; i < N3; i++ ){
		printf( "array[ %d ] = %d ", i, array[ i ] );
		if( (i+1) % 8 == 0 ){
			printf( "\n" );
		}
	}
	//**********end problem 4**********

//DO NOT COMMENT ANYTHING BELOW THIS LINE

	hipFree( d_array );
	free( array );

	return 0;
}
